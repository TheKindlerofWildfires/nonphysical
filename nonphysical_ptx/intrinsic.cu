#include<hip/hip_runtime.h>
#include <stdio.h>
//

extern "C" __global__ void test() {
    int laneId = threadIdx.x & 0x1f;
    // Seed starting value as inverse lane ID
    int value = 31 - laneId;

    // Use XOR mode to perform butterfly reduction
    for (int i=16; i>=1; i/=2)
        value += __shfl_xor_sync(0xffffffff, value, i, 32);

    // "value" now contains the sum across all threads
    float z = 1.0;
    if (value>1){
        z = NAN;
    }
    if (isnan(z)){
        printf("Thread %d final value = %d\n", threadIdx.x, value);
    }

}
int main() {
    test<<< 1, 32 >>>();
    hipDeviceSynchronize();

    return 0;
}