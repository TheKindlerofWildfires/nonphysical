#include<hip/hip_runtime.h>

//

__shared__ int block[4096];

extern "C" __global__ void test(unsigned int * x,float * y, int z) {
    block[threadIdx.x]=x[0];
    atomicAdd(&block[threadIdx.x],block[threadIdx.y]);
    x[1]=block[threadIdx.x];


}

//idea -> only atomic operations on shared memory to be thread safe 